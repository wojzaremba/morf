#include "hip/hip_runtime.h"
#include <stdio.h>
#include <algorithm>
#include <vector>
#include "mex.h"

//#include "cudaconvmono2.cuh"
#include "filter_acts_mono.cuh"
#include "nvmatrix.cuh"
#include "nvmatrix_kernels.cuh"
#include "nvmatrix_operators.cuh"



using namespace std;

static std::vector<NVMatrix*> p;
static hipEvent_t start_event, end_event;


#define GET3()       assert_((nrhs == 4) && (nlhs == 0)); \
   	             NVMatrix* a = getMatrix(prhs[1]); \
	             NVMatrix* b = getMatrix(prhs[2]); \
	             NVMatrix* c = getMatrix(prhs[3]);

#define GET1X1(TYPE) assert_((nrhs == 4) && (nlhs == 0)); \
		     NVMatrix* a = getMatrix(prhs[1]); \
	             TYPE b = (TYPE)mxGetScalar(prhs[2]); \
	             NVMatrix* c = getMatrix(prhs[3]);
                 
#define GET2()       assert_((nrhs == 3) && (nlhs == 0)); \
	             NVMatrix* a = getMatrix(prhs[1]); \
	             NVMatrix* b = getMatrix(prhs[2]); 


NVMatrix* getMatrix(const mxArray *prhs) {
	int idx = (int)mxGetScalar(prhs);
	assert_(idx < p.size());
	assert_(p[idx] != NULL);
	return p[idx];
}

void cleanUp() {
	for (int i = 0; i < p.size(); ++i) {
		if (p[i] != NULL) {
			delete(p[i]);
		}
	}
	p.clear();
	hipDeviceReset();
}

void CleanGPU(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	assert_(nrhs == 1 && nlhs == 0);
	cleanUp();
}

void ConvActMono(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	assert_((nrhs == 10) && (nlhs == 0));
	NVMatrix* images = getMatrix(prhs[1]);
	NVMatrix* filters = getMatrix(prhs[2]); 
	NVMatrix* targets = getMatrix(prhs[3]); 

	const mwSize* images_dims = mxGetDimensions(prhs[1]);
	int imgSizeY = (int)mxGetScalar(prhs[4]);
	int paddingStart = (int)mxGetScalar(prhs[8]);

	int filterSize = (int)mxGetScalar(prhs[6]);
	int moduleStride = (int)mxGetScalar(prhs[7]);
	int numModulesY = 1 + int(ceil((2 * paddingStart + imgSizeY - filterSize) / float(moduleStride)));
	int numModulesX = numModulesY;
	int numImgColors = (int)mxGetScalar(prhs[5]);


	NVMatrix* perm = getMatrix(prhs[9]);

	images->transpose();
	filters->transpose();
	targets->transpose();
	convFilterActsMono(*images, *filters, *targets,
                       imgSizeY, numModulesY, numModulesX, -paddingStart, moduleStride,
                       numImgColors, 0, 1, *perm);
	images->transpose();
	filters->transpose();
	targets->transpose();
}

void Reshape(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
        assert_((nrhs == 4) && (nlhs == 0));
        NVMatrix* a = getMatrix(prhs[1]);
	int b = (int)mxGetScalar(prhs[2]);
	int c = (int)mxGetScalar(prhs[3]);
	a->reshape(c, b);
}

void ActEXP(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET2();
        a->apply(NVMatrixOps::Exp(), *b);
}

void ActRELU(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET2();
	a->maxWithScalar(0, *b); 
}

void dActRELU(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET2();
	a->biggerThanScalar(0, *b); 
}

void dActLINEAR(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET2();
	b = a;
}

void ActLINEAR(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET2();
	b = a;
}

void AddVector(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET3();
	a->addVector(*b, *c);
}

void Add(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET3();
	a->add(*b, *c);
}

void Subtract(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET3();
	a->subtract(*b, *c);
}

void Mult(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET1X1(float);
	a->rightMult(b, *c);
}

void MultM(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET3();
	a->rightMult(*b, *c);
}

void EltwiseMult(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET3();
	a->eltwiseMult(*b, *c);
}

void Sum(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET1X1(int);
	a->sum(b, *c);
}

void Max(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET1X1(int);
	a->max(b, *c);
}

void Transpose(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	assert_((nrhs == 2) && (nlhs == 0));
	NVMatrix* a = getMatrix(prhs[1]);
	a->transpose();
}

void EltwiseDivideByVector(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	GET2();
	a->eltwiseDivideByVector(*b);
}

void PrintShape(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	assert_(nrhs == 2 && nlhs == 0);
	NVMatrix* nvmatrix = getMatrix(prhs[1]);
	nvmatrix->printShape("matrix");
}

void CopyFromGPU(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	assert_(nrhs == 2 && nlhs == 1);
	NVMatrix* nvmatrix = getMatrix(prhs[1]);
	plhs[0] = nvmatrix->copyToHost();
}

void CopyToGPU(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	assert_((nrhs == 3) && (nlhs == 0));
	int pid = (int)mxGetScalar(prhs[1]);
	if (p.size() <= pid) {
		for (int i = p.size(); i <= pid; ++i) {
			p.push_back(NULL);
		}
	}
	if (p[pid] != NULL) {
		p[pid]->copyFromHost(prhs[2], true);
	} else { 
		p[pid] = new NVMatrix(prhs[2], true);
	}
	p[pid]->setTrans(true);
}

void StartTimer(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	assert_(nrhs == 1 && nlhs == 0);
	hipEventCreate(&start_event);
	hipEventCreate(&end_event);
	hipEventRecord(start_event, 0);
}

void StopTimer(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	assert_(nrhs == 1 && nlhs == 1);
	hipEventRecord(end_event, 0);
	hipEventSynchronize(start_event);
	hipEventSynchronize(end_event);
    	plhs[0] = mxCreateNumericMatrix(1, 1, mxSINGLE_CLASS, mxREAL);
	float* lapse = (float*) mxGetData(plhs[0]);
	hipEventElapsedTime(lapse, start_event, end_event);
	(*lapse) /= 1000.; // returned time is in ms.
}

const int fsize = 26;
static void (*func[fsize]) (int, mxArray **, int, const mxArray **) = 
	{CopyToGPU, CopyFromGPU, AddVector, Mult, 
	 ActRELU, dActRELU, ActLINEAR, dActLINEAR,
	 NULL, Reshape, NULL, PrintShape, ActEXP,
	 Sum, Max, EltwiseDivideByVector, MultM, NULL, CleanGPU,
	 StartTimer, StopTimer, EltwiseMult, Transpose, Add, Subtract, ConvActMono};

void mexFunction (int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	assert_(nrhs >= 1);
	mexAtExit(cleanUp);
	int fid = (int)mxGetScalar(prhs[0]);
	assert_(fid < fsize);
	(*func[fid])(nlhs, plhs, nrhs, prhs);
}

